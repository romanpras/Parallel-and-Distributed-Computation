#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : FinalProject.c
 Author      : Roman Prasolov id- 313091746
 Version     :
 Copyright   : Your copyright notice
 Description : Cuda Functions
 ============================================================================
 */
 
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>

#include "cudaFunctions.h"
#include "cFunction.h"

// checks if str1 and str2 belong to the same conservative group
__device__ void groupCheck(const char *conservative[], int size, char result[], int index, char str1, char str2, char sign) {
	int group1, group2;
	for (int j = 0; j < size; j++) { //size of groups array
		group1 = -1;
		group2 = -1;
		for (int k = 0; conservative[j][k] != '\0'; k++) {
			if (group1 == -1 && str1 == conservative[j][k])
				group1 = j;
			if (group2 == -1 && str2 == conservative[j][k])
				group2 = j;
			if (group1 != -1 && group1 == group2) {
				result[index] = sign;
				return;
			}
		}
	}
}

// compare pair in semi-conservative groups
__device__ void semiConservativeGroupAction(char str1, char str2, int index, char result[]) {
	const char *semiConservative[11] = { "SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM" };
	groupCheck(semiConservative, 11, result, index, str1,str2, '.');
}

// compare pair in conservative groups
__device__ void conservativeGroupAction(char str1, char str2, int index, char result[]) {
	const char *conservative[9] = { "NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF" };
	groupCheck(conservative, 9, result, index, str1, str2,':');
}

// compare 2 sequences with offset
__global__ void compareStringsWithOffset(char *str1, char *str2, char *result, int size, int offset) {
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		result[index] = ' ';
		if (str2[index] == '-')
			return;
		if (str1[index + offset] == str2[index])
			result[index] = '*';
		else {
			conservativeGroupAction(str1[index + offset], str2[index], index, result);
			if (result[index] != ':')
				semiConservativeGroupAction(str1[index + offset], str2[index], index,result);
		}
	}
}

// allocate device memory
char* mallocForStr(size_t sizeT){
	char *str;
	hipError_t err = hipMalloc((void **)&str, sizeT);
	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    }
	return str;
}

// copy str to memory according to direction 0
void memCopyForStr(size_t sizeT,char *str, char *data, int direction){
	hipError_t err;
	if (direction==1)
		err = hipMemcpy(str, data, sizeT, hipMemcpyDeviceToHost);
	else 
		err = hipMemcpy(str, data, sizeT, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from %s - %s\n", (direction==0 ? "host to device" : "device to host"), hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


// return best result as double* of all opportunities of offsets.
double* computeOnGPU(char str1[], char str2[], int size,int mutant, int startOffset, int endOffset, double* weight) {

	size_t sizeT = size; 
	char *d_str1,*d_str2,*d_result, *result;
	
	// 0 - hipMemcpyHostToDevice
	int direction=0; 

	double *best = (double*) malloc(sizeof(double)*3);
	if (best==NULL) {
		printf("Failed to allocate the best array memory\n");
		exit(1);
	}

	// best offset
	best[0] = 0;
	
	// best mutant
	best[1] = mutant;
	
	// best score
	best[2] = -INFINITY;
	
	// allocate memory on GPU to copy the data from the host
	d_str1=mallocForStr(strlen(str1));
	d_str2=mallocForStr(sizeT);
	d_result=mallocForStr(sizeT);
	
	// allocate memory for the result 
	result = (char*)malloc(sizeof(char)*sizeT);
	if (!result) {
		fprintf(stderr, "Failed to allocate host memory\n");
        	exit(EXIT_FAILURE);
	}
	
	// Copy data from host to the GPU memory 
	memCopyForStr(strlen(str1), d_str1, str1, direction);
	memCopyForStr(sizeT, d_str2, str2, direction);
	
	// Launch the Kernel: calculate number of threads and blocks 
	int threadsPerBlock;
	int blocksPerGrid;
	if (size <= 1024){
		threadsPerBlock = size;
	}
	else {
		threadsPerBlock = 1024;
	}
	blocksPerGrid=size/threadsPerBlock;
	if (size%threadsPerBlock!=0)
		blocksPerGrid++;
		
	// 1 - hipMemcpyDeviceToHost 
	direction=1; 
	double currentBest[2];
	
	// calculate thes best score of the all offsets (startOffset to endOffset) 
	for (int i = startOffset; i < endOffset; i++) {
		currentBest[0] = i;
		compareStringsWithOffset<<<blocksPerGrid,threadsPerBlock>>>(d_str1, d_str2, d_result, size, i);
		memCopyForStr(sizeT, result, d_result, direction);
		result[sizeT] = '\0';
		
		// calculate the score
		currentBest[1] = calculateScore(weight, result); 
		
		// sets the best score
		if (currentBest[1] > best[2]) {
			best[2] = currentBest[1];
			best[0] = currentBest[0];
		}
	}
	
	// free the memory
	hipFree(d_str1);
	hipFree(d_str2);
	hipFree(d_result);
	free(result);
    return best;
}

